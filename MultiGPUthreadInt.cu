#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <semaphore.h>

#include <hip/hip_runtime.h>

#include "simpleMultiGPU.h"

//const int DATA_N = 10000000;
const int DATA_N = 100;


void* gpu_monitor(void* arg);
__global__ void add(int* a, int n);

sem_t localGPU_TaskSizeSubmit_sem;     
sem_t localGPU_TaskSubmit_sem;         
sem_t localGPU_beginComputation_sem;   
sem_t localGPU_computationFin_sem;

typedef struct
{
    //Host-side input data and computation result,if computation is not int summary,typedef sth substitute int
    int taskNo;
	int result;
	void *ptBegin;   	//reserved   raw data poniter(if taskNo > 4     malloc)
	int *ptSize;		//reserved   raw data size poniter(if taskNo > 4     malloc)
    void *begin[4];  	//initial container for raw data pointer
	int size[4];		//initial container for raw data size pointer; in bytes
}	GPUTask;

GPUTask gpuTask;

int main()
{
	int res;
    pthread_t gpu_thread;
    void *thread_result;

	res = sem_init(&localGPU_TaskSizeSubmit_sem, 0, 0);
	if(res != 0)
	{
		perror("sem localGPU_TaskSubmit_sem initialization failed");
		exit(EXIT_FAILURE);
	}

	res = sem_init(&localGPU_TaskSubmit_sem, 0, 0);
	if(res != 0)
	{
		perror("sem localGPU_TaskSubmit_sem initialization failed");
		exit(EXIT_FAILURE);
	}
	res = sem_init(&localGPU_beginComputation_sem, 0, 0);
	if(res != 0)
	{
	perror("sem localGPU_beginComputation_sem initialization failed");
	exit(EXIT_FAILURE);
	}
	res = sem_init(&localGPU_computationFin_sem, 0, 0);
	if(res != 0)
	{
		perror("sem localGPU_computationFin_sem initialization failed");
		exit(EXIT_FAILURE);
	}
	res = pthread_create(&gpu_thread, NULL, gpu_monitor, NULL);
	if (res != 0)
	{
		perror("Thread creation failed");
		exit(EXIT_FAILURE);
	}

	//gpuTask.taskNo = 4;
	gpuTask.taskNo = 2;
	int i, j;
	
	for(i = 0; i < gpuTask.taskNo; i++)
	{
		gpuTask.size[i] = DATA_N * sizeof(int);
	}

	sem_post(&localGPU_TaskSizeSubmit_sem);

	sem_wait(&localGPU_TaskSubmit_sem);

	int * intArray;
	int cpuSum = 0;
	for(i = 0;i < gpuTask.taskNo; i++)
	{
		intArray = (int *)gpuTask.begin[i];
		for(j = 0; j < DATA_N; j++)
			cpuSum += intArray[j] = j;
	}
	sem_post(&localGPU_beginComputation_sem);
	sem_wait(&localGPU_computationFin_sem);
	printf("The sum of gpu is %d\n", gpuTask.result);
	printf("The sum of cpu is %d\n", cpuSum);

    res = pthread_join(gpu_thread, &thread_result);
    if (res != 0) {
        perror("Thread join failed");
        exit(EXIT_FAILURE);
    }
    printf("Thread joined\n");
 
	sem_destroy(&localGPU_TaskSizeSubmit_sem);
	sem_destroy(&localGPU_TaskSubmit_sem);
	sem_destroy(&localGPU_beginComputation_sem);
	sem_destroy(&localGPU_computationFin_sem);
    exit(EXIT_SUCCESS);	

	return 0;
}

void *gpu_monitor(void *arg) {
	sem_wait(&localGPU_TaskSizeSubmit_sem);  // P²Ù×÷
	
	//int GPU_num = 0;
	//hipGetDeviceCount(&GPU_num);
	//printf("The count of GPU is %d\n", GPU_num);	
	
	TGPUplan plan[4];
	int i, j;
	
	for(i = 0; i < gpuTask.taskNo; i++)
	{
		gpuTask.begin[i] = malloc(gpuTask.size[i]);
		hipSetDevice(i);
		hipStreamCreate(&plan[i].stream);
		hipMalloc((void**)&plan[i].d_Data, gpuTask.size[i]);
		//hipHostMalloc((void**)&plan[i].h_Sum, sizeof(int));
		hipHostMalloc((void**)&plan[i].h_Data, gpuTask.size[i]);

		plan[i].h_Sum = (int *)malloc(sizeof(int));

		printf("GPU %d hipMalloc success.\n", i);

	}
	
	sem_post(&localGPU_TaskSubmit_sem);   // V²Ù×÷
	
	sem_wait(&localGPU_beginComputation_sem);	
	
	int sumGPUs = 0;
	int *intArray;

	for(i = 0; i < gpuTask.taskNo; i++)
	{
		intArray = (int* )gpuTask.begin[i];
		for(j = 0; j < DATA_N; j++)
			plan[i].h_Data[j] = intArray[j];
	}

	for(i = 0; i < gpuTask.taskNo; i++)
	{
		hipSetDevice(i);
		hipMemcpyAsync(plan[i].d_Data, plan[i].h_Data, gpuTask.size[i], hipMemcpyHostToDevice, plan[i].stream);
		printf("hipMemcpy success.\n");

		add_kernel<<<1, 2, 0, plan[i].stream>>>(plan[i].d_Data, DATA_N);

		hipMemcpyAsync(plan[i].h_Sum, plan[i].d_Data, sizeof(int), hipMemcpyDeviceToHost, plan[i].stream);
		sumGPUs += plan[i].h_Sum[0];
		printf("sum: %d\n", plan[i].h_Sum[0]);

		hipFree(plan[i].d_Data);
		hipHostFree(plan[i].h_Data);
		hipStreamDestroy(plan[i].stream);
	}

	gpuTask.result = sumGPUs;
	sem_post(&localGPU_computationFin_sem);

	return 0;
}
