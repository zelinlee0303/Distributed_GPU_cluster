#include <stdio.h>
#include <unistd.h>
#include <stdlib.h>
#include <string.h>
#include <pthread.h>
#include <semaphore.h>

#include <hip/hip_runtime.h>


//const int DATA_N = 10000000;
const int DATA_N = 100;


void* gpu_monitor(void* arg);
__global__ void add(int* a, int n);

sem_t localGPU_TaskSizeSubmit_sem;     
sem_t localGPU_TaskSubmit_sem;         
sem_t localGPU_beginComputation_sem;   
sem_t localGPU_computationFin_sem;

typedef struct
{
    //Host-side input data and computation result,if computation is not int summary,typedef sth substitute int
    int taskNo;
	int result;
	void *ptBegin;   	//reserved   raw data poniter(if taskNo > 4     malloc)
	int *ptSize;		//reserved   raw data size poniter(if taskNo > 4     malloc)
    void *begin[4];  	//initial container for raw data pointer
	int size[4];		//initial container for raw data size pointer; in bytes
}	GPUTask;

GPUTask gpuTask;

int main()
{
	int res;
    pthread_t gpu_thread;
    void *thread_result;

	res = sem_init(&localGPU_TaskSizeSubmit_sem, 0, 0);
	if(res != 0)
	{
		perror("sem localGPU_TaskSubmit_sem initialization failed");
		exit(EXIT_FAILURE);
	}

	res = sem_init(&localGPU_TaskSubmit_sem, 0, 0);
	if(res != 0)
	{
		perror("sem localGPU_TaskSubmit_sem initialization failed");
		exit(EXIT_FAILURE);
	}
	res = sem_init(&localGPU_beginComputation_sem, 0, 0);
	if(res != 0)
	{
	perror("sem localGPU_beginComputation_sem initialization failed");
	exit(EXIT_FAILURE);
	}
	res = sem_init(&localGPU_computationFin_sem, 0, 0);
	if(res != 0)
	{
		perror("sem localGPU_computationFin_sem initialization failed");
		exit(EXIT_FAILURE);
	}
	res = pthread_create(&gpu_thread, NULL, gpu_monitor, NULL);
	if (res != 0)
	{
		perror("Thread creation failed");
		exit(EXIT_FAILURE);
	}

	//gpuTask.taskNo = 4;
	gpuTask.taskNo = 1;
	int i, j;
	
	for(i = 0; i < gpuTask.taskNo; i++)
	{
		gpuTask.size[i] = DATA_N * sizeof(int);
	}

	sem_post(&localGPU_TaskSizeSubmit_sem);

	sem_wait(&localGPU_TaskSubmit_sem);

	int * intArray;
	int cpuSum = 0;
	for(i = 0;i < gpuTask.taskNo; i++)
	{
		intArray = (int *)gpuTask.begin[i];
		//ÆäÊµÕâÀïÓ¦¸ÃÅÐ¶ÏDATA_N*sizeof(int)ÊÇ²»ÊÇÓësize[i]ÏàµÈ~
		for(j = 0; j < DATA_N; j++)
			cpuSum += intArray[j] = j;
	}
	sem_post(&localGPU_beginComputation_sem);
	sem_wait(&localGPU_computationFin_sem);
	printf("The sum of gpu is %d\n",gpuTask.result);
	printf("The sum of cpu is %d\n",cpuSum);

    res = pthread_join(gpu_thread, &thread_result);
    if (res != 0) {
        perror("Thread join failed");
        exit(EXIT_FAILURE);
    }
    printf("Thread joined\n");
 
	sem_destroy(&localGPU_TaskSizeSubmit_sem);
	sem_destroy(&localGPU_TaskSubmit_sem);
	sem_destroy(&localGPU_beginComputation_sem);
	sem_destroy(&localGPU_computationFin_sem);
    exit(EXIT_SUCCESS);	

	return 0;
}

void *gpu_monitor(void *arg) {
	sem_wait(&localGPU_TaskSizeSubmit_sem);  // P²Ù×÷
	
	//int GPU_num = 0;
	//cudaGetDeviceCount(&GPU_num);
	//printf("The count of GPU is %d\n", GPU_num);	
	
	int i;
	int* mission; //data on gpu

	for(i = 0; i < gpuTask.taskNo; i++)
	{
		gpuTask.begin[i] = malloc(gpuTask.size[i]);
		hipSetDevice(2);
		hipMalloc((void**)&mission, gpuTask.size[i]);
		//cudaMemcpy(mission[i], &intArray[0], gpuTask.size[i], cudaMemcpyHostToDevice);
		printf("GPU %d hipMalloc success.\n", i);
	}
	sem_post(&localGPU_TaskSubmit_sem);   // V²Ù×÷
	
	sem_wait(&localGPU_beginComputation_sem);	
	
	int* intArray;
	int sumGPUs = 0;
	int* sum;
	for(i = 0; i < gpuTask.taskNo; i++)
	{
		//sum = 0;
		intArray = (int *)gpuTask.begin[i];
		//cudaSetDevice(i);
		hipMemcpy(mission, &intArray[0], gpuTask.size[i], hipMemcpyHostToDevice);
		printf("hipMemcpy success.\n");
		add<<<1, 2>>>(mission, DATA_N);
		//for(j = 0; j < DATA_N; j++)
			//sum += intArray[j];
		hipMemcpy(sum, mission, sizeof(int), hipMemcpyDeviceToHost);
		sumGPUs += sum[0];
		printf("ok\n");
		printf("sum: %d\n", sum[0]);
	}
	gpuTask.result = sumGPUs;
	sem_post(&localGPU_computationFin_sem);
}

__global__ void add(int* a, int n)
{
	int i = threadIdx.x+blockDim.x+blockIdx.x;

	for(i = 1; i < n; i++)
	{
		a[0] += a[i];
	}
}
